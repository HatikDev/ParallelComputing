#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Constants.h"
#include "Generator.h"
#include "Serialization.h"

#include <sstream>
#include <chrono>

#include <mpi.h>

using std::chrono::steady_clock;

namespace
{
__device__ int dev_maxPower(uint16_t number)
{
    int powerArray = 16384;
    for (int i = 7; i >= 0; --i)
    {
        if (number & powerArray)
            return i + 8;
        powerArray >= 1;
    }

    return 0;
}
}

__device__ uint8_t dev_plus(uint8_t num1, uint8_t num2)
{
    return num1 ^ num2;
}

__device__ uint8_t dev_mult(uint8_t num1, uint8_t num2)
{
    uint16_t result = 0;

    for (int i = 0; i < 8; ++i)
    {
        if (!(num1 & 1))
        {
            num1 >>= 1;
            continue;
        }

        uint8_t num2_2 = num2;
        for (int j = 0; j < 8; ++j)
        {
            if (!(num2_2 & 1))
            {
                num2_2 >>= 1;
                continue;
            }

            result ^= (1 << (i + j));
            num2_2 >>= 1;
        }

        num1 >>= 1;
    }

    while (result > 255)
    {
        int power = dev_maxPower(result);
        result ^= 0x11b << (power - 8);
    }

    return static_cast<uint8_t>(result);
}

__global__ void transformMatrix(uint8_t* coeffs, uint8_t* dev_inMatrix, uint8_t* dev_outMatrix)
{
    int index = threadIdx.x;
    uint8_t* inMatrixBegin = dev_inMatrix + index * M * M;
    uint8_t* outMatrixBegin = dev_outMatrix + index * M * M;

    uint8_t sum = 0;
    for (size_t i = 0; i < M; ++i)
    {
        for (size_t j = 0; j < M; ++j)
        {
            sum = 0;
            for (size_t k = 0; k < M; ++k)
            {
                sum += dev_plus(sum, dev_mult(coeffs[i * M + k], dev_inMatrix[k * M + j]));
            }
            dev_outMatrix[i * M + j] = sum;
        }
    }
}

hipError_t initCoeffsMatrix(uint8_t** coeffs)
{
    hipError_t cudaStatus = hipMalloc((void**)coeffs, 16 * sizeof(uint8_t));
    if (cudaStatus != hipSuccess)
        return cudaStatus;

    uint8_t hipArray[] = { 2, 3, 1, 1, 1, 2, 3, 1, 1, 1, 2, 3, 3, 1, 1, 2 };
    cudaStatus = hipMemcpy(*coeffs, hipArray, 16, hipMemcpyHostToDevice);

    return cudaStatus;
}

hipError_t cudaTransformData(size_t N, std::string inFileName, std::string outFileName)
{
    VecMatrix inMatrixes(N, Matrix());
    VecMatrix outMatrixes(N, Matrix());

    FileReader reader(inFileName);
    FileWriter writer(outFileName);

    reader.startReading();
    reader.readData(inMatrixes.begin(), inMatrixes.end());
    reader.finishReading();

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // create coeffs for multiplying matrixes

    uint8_t* dev_coeffs;
    cudaStatus = hipMalloc((void**)&dev_coeffs, 16 * sizeof(uint8_t));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc for coeffs failed!");
        return cudaStatus;
    }

    uint8_t hipArray[] = { 2, 3, 1, 1, 1, 2, 3, 1, 1, 1, 2, 3, 3, 1, 1, 2 };
    cudaStatus = hipMemcpy(dev_coeffs, hipArray, 16, hipMemcpyHostToDevice);

    // prepare cuda memory for input matrixes

    uint8_t* dev_inMatrixes;
    cudaStatus = hipMalloc((void**)&dev_inMatrixes, M * M * N * sizeof(uint8_t));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc for input matrixes failed!");
        goto Error;
    }

    // prepare cuda memory for output matrixes

    uint8_t* dev_outMatrixes;
    cudaStatus = hipMalloc((void**)&dev_outMatrixes, M * M * N * sizeof(uint8_t));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc for output matrixes failed!");
        goto Error;
    }

    // copy input matrixes into host memory

    uint8_t* matrixesCopy = (uint8_t*)malloc(M * M * N);
    uint8_t* pointer = matrixesCopy;
    for (auto& matrix : inMatrixes)
    {
        for (size_t i = 0; i < M; ++i)
        {
            for (size_t j = 0; j < M; ++j)
            {
                *pointer = matrix[i][j];
                ++pointer;
            }
        }
    }

    // copy input matrixes into cuda memory

    cudaStatus = hipMemcpy(dev_inMatrixes, matrixesCopy, M * M * N * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy matrixes failed!");
        goto Error;
    }

    // transform matrixes

    hipEvent_t start;
    hipEvent_t stop;
    float duration;

    hipEventCreate(&start);
    hipEventRecord(start);

    transformMatrix<<<1, N>>>(dev_coeffs, dev_inMatrixes, dev_outMatrixes);

    hipEventCreate(&stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&duration, start, stop);

    // copy output matrixes from cuda into host memory

    cudaStatus = hipMemcpy(matrixesCopy, dev_inMatrixes, M * M * N * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy matrixes failed!");
        goto Error;
    }

    // copy output matrixes into std object

    pointer = matrixesCopy;
    for (auto& matrix : outMatrixes)
    {
        for (size_t i = 0; i < M; ++i)
        {
            for (size_t j = 0; j < M; ++j)
            {
                matrix[i][j] = *pointer;
                ++pointer;
            }
        }
    }

    // write data in file

    writer.startWriting();
    writer.writeData(outMatrixes.begin(), outMatrixes.end());
    writer.finishWriting();

    MPI_Send(&duration, 1, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);

Error:
    hipFree(dev_coeffs);
    hipFree(dev_inMatrixes);
    hipFree(dev_outMatrixes);
    free(matrixesCopy);

    return cudaStatus;
}
